
#include <hip/hip_runtime.h>
extern "C" __device__ void begin_parallel_region();
extern "C" __device__ void end_parallel_region();

extern "C" __device__ void kernel(int n, int* inputs, int* outputs)
{
	outputs[0] = 0;

	begin_parallel_region();
	for (int i = 1; i < n - 1; i++)
		if (inputs[i] < 2)
			outputs[i] = inputs[i] + i;
		else
			outputs[i] = inputs[i];
	end_parallel_region();

	outputs[n - 1] = n - 1;
}
